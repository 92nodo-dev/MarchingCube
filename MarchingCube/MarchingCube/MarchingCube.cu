#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "MarchingCube.cuh"

__global__ void compute_bit(Cell* cell, int x, int y, int z, float isoValue) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int xIndex = int(idx / ((z + 1) * (y + 1)));
	int yIndex = int(idx % ((z + 1) * (y + 1))) / (z + 1);
	int zIndex = int(idx % ((z + 1) * (y + 1))) % (z + 1);

	if ((xIndex % x != 0) && (yIndex % y != 0) && (zIndex % z != 0))
	{
		float avgDensity = 0;

		avgDensity += cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex - 1)].density;
		avgDensity += cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex)].density;
		avgDensity += cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex)].density;
		avgDensity += cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex - 1)].density;
		avgDensity += cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex - 1)].density;
		avgDensity += cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex)].density;
		avgDensity += cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex)].density;
		avgDensity += cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex - 1)].density;

		avgDensity *= 0.125;

		cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex - 1)].valueOfVertex[6] = avgDensity;
		cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex)].valueOfVertex[2] = avgDensity;
		cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex)].valueOfVertex[1] = avgDensity;
		cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex - 1)].valueOfVertex[5] = avgDensity;
		cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex - 1)].valueOfVertex[7] = avgDensity;
		cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex)].valueOfVertex[3] = avgDensity;
		cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex)].valueOfVertex[0] = avgDensity;
		cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex - 1)].valueOfVertex[4] = avgDensity;

		if (avgDensity < isoValue)
		{
			//printf("idx : %d\n", idx);
			cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex - 1)].isUsingVertex[6] = true;
			cell[(z * y * (xIndex - 1)) + (z * (yIndex - 1)) + (zIndex)].isUsingVertex[2] = true;
			cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex)].isUsingVertex[1] = true;
			cell[(z * y * (xIndex - 1)) + (z * (yIndex)) + (zIndex - 1)].isUsingVertex[5] = true;
			cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex - 1)].isUsingVertex[7] = true;
			cell[(z * y * (xIndex)) + (z * (yIndex - 1)) + (zIndex)].isUsingVertex[3] = true;
			cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex)].isUsingVertex[0] = true;
			cell[(z * y * (xIndex)) + (z * (yIndex)) + (zIndex - 1)].isUsingVertex[4] = true;
		}
	}
}

__global__ void make_cell_triangle(Cell* cell, int* d_edgeTable, short int* d_triTable, int x, int y, int z, float isoValue) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int usage = 0;

	if (cell[idx].isUsingVertex[0]) usage += 1;
	if (cell[idx].isUsingVertex[1]) usage += 2;
	if (cell[idx].isUsingVertex[2]) usage += 4;
	if (cell[idx].isUsingVertex[3]) usage += 8;
	if (cell[idx].isUsingVertex[4]) usage += 16;
	if (cell[idx].isUsingVertex[5]) usage += 32;
	if (cell[idx].isUsingVertex[6]) usage += 64;
	if (cell[idx].isUsingVertex[7]) usage += 128;

	int usingEdge = d_edgeTable[usage];

	// max = v2, min = v1
	
	float k1, k2, v1, v2;
	vec3 p1, p2;
	if (usingEdge & 1)
	{
		v1 = cell[idx].valueOfVertex[1] < cell[idx].valueOfVertex[0] ? cell[idx].valueOfVertex[1] : cell[idx].valueOfVertex[0];
		v2 = cell[idx].valueOfVertex[1] > cell[idx].valueOfVertex[0] ? cell[idx].valueOfVertex[1] : cell[idx].valueOfVertex[0];
		p1 = cell[idx].valueOfVertex[1] < cell[idx].valueOfVertex[0] ? cell[idx].vertex[1] : cell[idx].vertex[0];
		p2 = cell[idx].valueOfVertex[1] > cell[idx].valueOfVertex[0] ? cell[idx].vertex[1] : cell[idx].vertex[0];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[0] = (cell[idx].vertex[0] + cell[idx].vertex[1]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[0] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 2)
	{
		v1 = cell[idx].valueOfVertex[2] < cell[idx].valueOfVertex[1] ? cell[idx].valueOfVertex[2] : cell[idx].valueOfVertex[1];
		v2 = cell[idx].valueOfVertex[2] > cell[idx].valueOfVertex[1] ? cell[idx].valueOfVertex[2] : cell[idx].valueOfVertex[1];
		p1 = cell[idx].valueOfVertex[2] < cell[idx].valueOfVertex[1] ? cell[idx].vertex[2] : cell[idx].vertex[1];
		p2 = cell[idx].valueOfVertex[2] > cell[idx].valueOfVertex[1] ? cell[idx].vertex[2] : cell[idx].vertex[1];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[1] = (cell[idx].vertex[1] + cell[idx].vertex[2]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[1] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 4)
	{
		v1 = cell[idx].valueOfVertex[3] < cell[idx].valueOfVertex[2] ? cell[idx].valueOfVertex[3] : cell[idx].valueOfVertex[2];
		v2 = cell[idx].valueOfVertex[3] > cell[idx].valueOfVertex[2] ? cell[idx].valueOfVertex[3] : cell[idx].valueOfVertex[2];
		p1 = cell[idx].valueOfVertex[3] < cell[idx].valueOfVertex[2] ? cell[idx].vertex[3] : cell[idx].vertex[2];
		p2 = cell[idx].valueOfVertex[3] > cell[idx].valueOfVertex[2] ? cell[idx].vertex[3] : cell[idx].vertex[2];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[2] = (cell[idx].vertex[3] + cell[idx].vertex[2]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[2] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 8)
	{
		v1 = cell[idx].valueOfVertex[0] < cell[idx].valueOfVertex[3] ? cell[idx].valueOfVertex[0] : cell[idx].valueOfVertex[3];
		v2 = cell[idx].valueOfVertex[0] > cell[idx].valueOfVertex[3] ? cell[idx].valueOfVertex[0] : cell[idx].valueOfVertex[3];
		p1 = cell[idx].valueOfVertex[0] < cell[idx].valueOfVertex[3] ? cell[idx].vertex[0] : cell[idx].vertex[3];
		p2 = cell[idx].valueOfVertex[0] > cell[idx].valueOfVertex[3] ? cell[idx].vertex[0] : cell[idx].vertex[3];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[3] = (cell[idx].vertex[0] + cell[idx].vertex[3]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[3] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 16)
	{
		v1 = cell[idx].valueOfVertex[4] < cell[idx].valueOfVertex[5] ? cell[idx].valueOfVertex[4] : cell[idx].valueOfVertex[5];
		v2 = cell[idx].valueOfVertex[4] > cell[idx].valueOfVertex[5] ? cell[idx].valueOfVertex[4] : cell[idx].valueOfVertex[5];
		p1 = cell[idx].valueOfVertex[4] < cell[idx].valueOfVertex[5] ? cell[idx].vertex[4] : cell[idx].vertex[5];
		p2 = cell[idx].valueOfVertex[4] > cell[idx].valueOfVertex[5] ? cell[idx].vertex[4] : cell[idx].vertex[5];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[4] = (cell[idx].vertex[4] + cell[idx].vertex[5]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[4] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 32)
	{
		v1 = cell[idx].valueOfVertex[5] < cell[idx].valueOfVertex[6] ? cell[idx].valueOfVertex[5] : cell[idx].valueOfVertex[6];
		v2 = cell[idx].valueOfVertex[5] > cell[idx].valueOfVertex[6] ? cell[idx].valueOfVertex[5] : cell[idx].valueOfVertex[6];
		p1 = cell[idx].valueOfVertex[5] < cell[idx].valueOfVertex[6] ? cell[idx].vertex[5] : cell[idx].vertex[6];
		p2 = cell[idx].valueOfVertex[5] > cell[idx].valueOfVertex[6] ? cell[idx].vertex[5] : cell[idx].vertex[6];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[5] = (cell[idx].vertex[5] + cell[idx].vertex[6]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[5] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 64)
	{
		v1 = cell[idx].valueOfVertex[6] < cell[idx].valueOfVertex[7] ? cell[idx].valueOfVertex[6] : cell[idx].valueOfVertex[7];
		v2 = cell[idx].valueOfVertex[6] > cell[idx].valueOfVertex[7] ? cell[idx].valueOfVertex[6] : cell[idx].valueOfVertex[7];
		p1 = cell[idx].valueOfVertex[6] < cell[idx].valueOfVertex[7] ? cell[idx].vertex[6] : cell[idx].vertex[7];
		p2 = cell[idx].valueOfVertex[6] > cell[idx].valueOfVertex[7] ? cell[idx].vertex[6] : cell[idx].vertex[7];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[6] = (cell[idx].vertex[6] + cell[idx].vertex[7]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[6] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 128)
	{
		v1 = cell[idx].valueOfVertex[7] < cell[idx].valueOfVertex[4] ? cell[idx].valueOfVertex[7] : cell[idx].valueOfVertex[4];
		v2 = cell[idx].valueOfVertex[7] > cell[idx].valueOfVertex[4] ? cell[idx].valueOfVertex[7] : cell[idx].valueOfVertex[4];
		p1 = cell[idx].valueOfVertex[7] < cell[idx].valueOfVertex[4] ? cell[idx].vertex[7] : cell[idx].vertex[4];
		p2 = cell[idx].valueOfVertex[7] > cell[idx].valueOfVertex[4] ? cell[idx].vertex[7] : cell[idx].vertex[4];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[7] = (cell[idx].vertex[7] + cell[idx].vertex[4]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[7] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 256)
	{
		v1 = cell[idx].valueOfVertex[0] < cell[idx].valueOfVertex[4] ? cell[idx].valueOfVertex[0] : cell[idx].valueOfVertex[4];
		v2 = cell[idx].valueOfVertex[0] > cell[idx].valueOfVertex[4] ? cell[idx].valueOfVertex[0] : cell[idx].valueOfVertex[4];
		p1 = cell[idx].valueOfVertex[0] < cell[idx].valueOfVertex[4] ? cell[idx].vertex[0] : cell[idx].vertex[4];
		p2 = cell[idx].valueOfVertex[0] > cell[idx].valueOfVertex[4] ? cell[idx].vertex[0] : cell[idx].vertex[4];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[8] = (cell[idx].vertex[0] + cell[idx].vertex[4]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[8] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 512)
	{
		v1 = cell[idx].valueOfVertex[1] < cell[idx].valueOfVertex[5] ? cell[idx].valueOfVertex[1] : cell[idx].valueOfVertex[5];
		v2 = cell[idx].valueOfVertex[1] > cell[idx].valueOfVertex[5] ? cell[idx].valueOfVertex[1] : cell[idx].valueOfVertex[5];
		p1 = cell[idx].valueOfVertex[1] < cell[idx].valueOfVertex[5] ? cell[idx].vertex[1] : cell[idx].vertex[5];
		p2 = cell[idx].valueOfVertex[1] > cell[idx].valueOfVertex[5] ? cell[idx].vertex[1] : cell[idx].vertex[5];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[9] = (cell[idx].vertex[1] + cell[idx].vertex[5]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[9] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 1024)
	{
		v1 = cell[idx].valueOfVertex[2] < cell[idx].valueOfVertex[6] ? cell[idx].valueOfVertex[2] : cell[idx].valueOfVertex[6];
		v2 = cell[idx].valueOfVertex[2] > cell[idx].valueOfVertex[6] ? cell[idx].valueOfVertex[2] : cell[idx].valueOfVertex[6];
		p1 = cell[idx].valueOfVertex[2] < cell[idx].valueOfVertex[6] ? cell[idx].vertex[2] : cell[idx].vertex[6];
		p2 = cell[idx].valueOfVertex[2] > cell[idx].valueOfVertex[6] ? cell[idx].vertex[2] : cell[idx].vertex[6];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[10] = (cell[idx].vertex[2] + cell[idx].vertex[6]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[10] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}
	if (usingEdge & 2048)
	{
		v1 = cell[idx].valueOfVertex[3] < cell[idx].valueOfVertex[7] ? cell[idx].valueOfVertex[3] : cell[idx].valueOfVertex[7];
		v2 = cell[idx].valueOfVertex[3] > cell[idx].valueOfVertex[7] ? cell[idx].valueOfVertex[3] : cell[idx].valueOfVertex[7];
		p1 = cell[idx].valueOfVertex[3] < cell[idx].valueOfVertex[7] ? cell[idx].vertex[3] : cell[idx].vertex[7];
		p2 = cell[idx].valueOfVertex[3] > cell[idx].valueOfVertex[7] ? cell[idx].vertex[3] : cell[idx].vertex[7];
		k1 = v2 - isoValue;
		k2 = isoValue - v1;
		if ((k1 <= 0) || (k2 <= 0)) {
			cell[idx].edgeVertex[11] = (cell[idx].vertex[3] + cell[idx].vertex[7]) * 0.5f;
			return;
		}
		cell[idx].edgeVertex[11] = ((p1 * k1) + (p2 * k2)) * (1 / (k1 + k2));
	}

/*
	if (usingEdge & 1)		cell[idx].edgeVertex[0] = cell[idx].vertex[0] +((cell[idx].vertex[1] - cell[idx].vertex[0]) * ((isoValue - cell[idx].valueOfVertex[0]) / (cell[idx].valueOfVertex[1] - cell[idx].valueOfVertex[0])));
	if (usingEdge & 2)		cell[idx].edgeVertex[1] = cell[idx].vertex[1] + ((cell[idx].vertex[2] - cell[idx].vertex[1]) * ((isoValue - cell[idx].valueOfVertex[1]) / (cell[idx].valueOfVertex[2] - cell[idx].valueOfVertex[1])));
	if (usingEdge & 4)		cell[idx].edgeVertex[2] = cell[idx].vertex[2] + ((cell[idx].vertex[3] - cell[idx].vertex[2]) * ((isoValue - cell[idx].valueOfVertex[2]) / (cell[idx].valueOfVertex[3] - cell[idx].valueOfVertex[2])));
	if (usingEdge & 8)		cell[idx].edgeVertex[3] = cell[idx].vertex[3] + ((cell[idx].vertex[0] - cell[idx].vertex[3]) * ((isoValue - cell[idx].valueOfVertex[3]) / (cell[idx].valueOfVertex[4] - cell[idx].valueOfVertex[3])));

	if (usingEdge & 16)		cell[idx].edgeVertex[4] = cell[idx].vertex[4] + ((cell[idx].vertex[5] - cell[idx].vertex[4]) * ((isoValue - cell[idx].valueOfVertex[4]) / (cell[idx].valueOfVertex[5] - cell[idx].valueOfVertex[4])));
	if (usingEdge & 32)		cell[idx].edgeVertex[5] = cell[idx].vertex[5] + ((cell[idx].vertex[6] - cell[idx].vertex[5]) * ((isoValue - cell[idx].valueOfVertex[5]) / (cell[idx].valueOfVertex[6] - cell[idx].valueOfVertex[5])));
	if (usingEdge & 64)		cell[idx].edgeVertex[6] = cell[idx].vertex[6] + ((cell[idx].vertex[7] - cell[idx].vertex[6]) * ((isoValue - cell[idx].valueOfVertex[6]) / (cell[idx].valueOfVertex[7] - cell[idx].valueOfVertex[6])));
	if (usingEdge & 128)	cell[idx].edgeVertex[7] = cell[idx].vertex[7] + ((cell[idx].vertex[4] - cell[idx].vertex[7]) * ((isoValue - cell[idx].valueOfVertex[7]) / (cell[idx].valueOfVertex[4] - cell[idx].valueOfVertex[7])));

	if (usingEdge & 256)	cell[idx].edgeVertex[8] = cell[idx].vertex[0] + ((cell[idx].vertex[4] - cell[idx].vertex[0]) * ((isoValue - cell[idx].valueOfVertex[0]) / (cell[idx].valueOfVertex[4] - cell[idx].valueOfVertex[0])));
	if (usingEdge & 512)	cell[idx].edgeVertex[9] = cell[idx].vertex[1] + ((cell[idx].vertex[5] - cell[idx].vertex[1]) * ((isoValue - cell[idx].valueOfVertex[1]) / (cell[idx].valueOfVertex[5] - cell[idx].valueOfVertex[1])));
	if (usingEdge & 1024)	cell[idx].edgeVertex[10] = cell[idx].vertex[2] + ((cell[idx].vertex[6] - cell[idx].vertex[2]) * ((isoValue - cell[idx].valueOfVertex[2]) / (cell[idx].valueOfVertex[6] - cell[idx].valueOfVertex[2])));
	if (usingEdge & 2048)	cell[idx].edgeVertex[11] = cell[idx].vertex[3] + ((cell[idx].vertex[7] - cell[idx].vertex[3]) * ((isoValue - cell[idx].valueOfVertex[3]) / (cell[idx].valueOfVertex[7] - cell[idx].valueOfVertex[3])));
*/

/*
	if (usingEdge & 1)		cell[idx].edgeVertex[0] = (cell[idx].vertex[0] + cell[idx].vertex[1]) * 0.5f;
	if (usingEdge & 2)		cell[idx].edgeVertex[1] = (cell[idx].vertex[1] + cell[idx].vertex[2]) * 0.5f;
	if (usingEdge & 4)		cell[idx].edgeVertex[2] = (cell[idx].vertex[2] + cell[idx].vertex[3]) * 0.5f; 
	if (usingEdge & 8)		cell[idx].edgeVertex[3] = (cell[idx].vertex[0] + cell[idx].vertex[3]) * 0.5f;

	if (usingEdge & 16)		cell[idx].edgeVertex[4] = (cell[idx].vertex[4] + cell[idx].vertex[5]) * 0.5f;
	if (usingEdge & 32)		cell[idx].edgeVertex[5] = (cell[idx].vertex[5] + cell[idx].vertex[6]) * 0.5f; 
	if (usingEdge & 64)		cell[idx].edgeVertex[6] = (cell[idx].vertex[6] + cell[idx].vertex[7]) * 0.5f;
	if (usingEdge & 128)	cell[idx].edgeVertex[7] = (cell[idx].vertex[7] + cell[idx].vertex[4]) * 0.5f;

	if (usingEdge & 256)	cell[idx].edgeVertex[8] = (cell[idx].vertex[0] + cell[idx].vertex[4]) * 0.5f; 
	if (usingEdge & 512)	cell[idx].edgeVertex[9] = (cell[idx].vertex[1] + cell[idx].vertex[5]) * 0.5f; 
	if (usingEdge & 1024)	cell[idx].edgeVertex[10] = (cell[idx].vertex[2] + cell[idx].vertex[6]) * 0.5f; 
	if (usingEdge & 2048)	cell[idx].edgeVertex[11] = (cell[idx].vertex[3] + cell[idx].vertex[7]) * 0.5f; 
*/

	for (int i = 0; i < 5; i++)
	{
		if (d_triTable[(usage * 16) + (i * 3)] == -1) {
			cell[idx].triangleCnt = i;
			break;
		}
		cell[idx].triangles[i].t1 = cell[idx].edgeVertex[d_triTable[(usage * 16) + (i * 3)]];
		cell[idx].triangles[i].t2 = cell[idx].edgeVertex[d_triTable[(usage * 16) + (i * 3) + 1]];
		cell[idx].triangles[i].t3 = cell[idx].edgeVertex[d_triTable[(usage * 16) + (i * 3) + 2]];

		if (d_triTable[(usage * 16) + (i * 3)] == 0) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[1]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 1) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[2]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 2) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[3]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 3) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[0]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 4) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[4] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 5) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[5] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 6) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[6] + cell[idx].valueOfVertex[7]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 7) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[7] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 8) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 9) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 10) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3)] == 11) cell[idx].triangles[i].density[0] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[7]) * 0.5f;

		if (d_triTable[(usage * 16) + (i * 3) + 1] == 0) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[1]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 1) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[2]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 2) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[3]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 3) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[0]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 4) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[4] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 5) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[5] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 6) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[6] + cell[idx].valueOfVertex[7]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 7) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[7] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 8) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 9) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 10) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 1] == 11) cell[idx].triangles[i].density[1] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[7]) * 0.5f;

		if (d_triTable[(usage * 16) + (i * 3) + 2] == 0) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[1]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 1) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[2]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 2) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[3]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 3) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[0]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 4) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[4] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 5) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[5] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 6) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[6] + cell[idx].valueOfVertex[7]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 7) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[7] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 8) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[0] + cell[idx].valueOfVertex[4]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 9) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[1] + cell[idx].valueOfVertex[5]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 10) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[2] + cell[idx].valueOfVertex[6]) * 0.5f;
		if (d_triTable[(usage * 16) + (i * 3) + 2] == 11) cell[idx].triangles[i].density[2] = (cell[idx].valueOfVertex[3] + cell[idx].valueOfVertex[7]) * 0.5f;
	}
}

__global__ void add_triangle_to_array(Cell* cell, Triangle* triangleArr) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int amount = cell[idx].triangleCnt;
	triangleArr[idx * 5 + 0] = cell[idx].triangles[0];
	triangleArr[idx * 5 + 1] = cell[idx].triangles[1];
	triangleArr[idx * 5 + 2] = cell[idx].triangles[2];
	triangleArr[idx * 5 + 3] = cell[idx].triangles[3];
	triangleArr[idx * 5 + 4] = cell[idx].triangles[4];
}

__global__ void interpolate_cell_density(Cell* cell) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
}

void MarchingCube::compute_cell_bit(float isoValue)
{
	compute_bit << <((axisX + 1) * (axisY + 1) * (axisZ + 1)) / 64, 64 >> > (d_data.cells, axisX, axisY, axisZ, isoValue);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA:ERROR:cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	}
	else {
		printf("CUDA Success\n");
	}
	printf("inside compute_cell_bit\n");
}

void MarchingCube::make_triangle(float isoValue)
{
	make_cell_triangle <<< ((axisX) * (axisY) * (axisZ)) / 64, 64 >> > (d_data.cells, d_data.edgeTable, d_data.triangleTable, axisX, axisY, axisZ, isoValue);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA:ERROR:cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	}
	else {
		printf("CUDA Success\n");
	}
	printf("inside make_cell_triangle\n");
}

void MarchingCube::make_triangle_arr()
{
/*
	int d_idx = 0;
	for (int i = 0; i < axisX; ++i)
	{
		for (int j = 0; j < axisY; ++j)
		{
			for (int k = 0; k < axisZ; ++k)
			{
				hipMemcpy(&(h_data.cells[i][j][k]), &(d_data.cells[d_idx]), sizeof(Cell), hipMemcpyDeviceToHost);
				d_idx++;
			}
		}
	}
*/

	printf("tssdf23423523523etset\n");
	hipMemcpy(h_data.cells, d_data.cells, sizeof(Cell)* axisX* axisY* axisZ, hipMemcpyDeviceToHost);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA:ERROR:cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	}
	else {
		printf("CUDA Success\n");
	}
	printf("tsetsetsetset\n");
	for (int i = 0; i < axisX; ++i)
	{
		for (int j = 0; j < axisY; ++j)
		{
			for (int k = 0; k < axisZ; ++k)
			{
				for (int l = 0; l < h_data.cells[(i*axisY*axisZ)+(j*axisZ)+k].triangleCnt; ++l)
				{
					h_data.triangles .push_back(h_data.cells[(i * axisY * axisZ) + (j * axisZ) + k].triangles[l]);
				}
			}
		}
	}
}

void MarchingCube::alloc_device_memory()
{
	int d_idx = 0;
	printf("inside alloc device memory\n");
	hipMalloc((void**)&(d_data.cells), axisX * axisY * axisZ * sizeof(Cell));
	hipMalloc((void**)&d_data.edgeTable, 256 * sizeof(int));
	hipMalloc((void**)&d_data.triangleTable, 256 * 16 * sizeof(short int));
	hipMalloc((void**)&d_data.triangles, axisX * axisY * axisZ * sizeof(Triangle));

	hipMemcpy(d_data.edgeTable, h_data.edgeTable, 256 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_data.triangleTable, h_data.triangleTable, 256 * 16 * sizeof(short int), hipMemcpyHostToDevice);
/*
	for (int i = 0; i < axisX; ++i)
	{
		for (int j = 0; j < axisY; ++j)
		{
			for (int k = 0; k < axisZ; ++k)
			{
				cells[i][j][k].index = d_idx;
				hipMemcpy(&d_cells[d_idx], &cells[i][j][k], sizeof(Cell), hipMemcpyHostToDevice);
				d_idx++;
			}
		}
	}    
*/

	hipMemcpy(d_data.cells, h_data.cells, sizeof(Cell) * axisX * axisY * axisZ, hipMemcpyHostToDevice);
	printf("inside alloc device memory\n");
}

void MarchingCube::free_device_memory()
{
	hipFree(d_data.cells);
	hipFree(d_data.edgeTable);
	hipFree(d_data.triangleTable);
	hipFree(d_data.triangles);
}

bool MarchingCube::get_vertices_by_txt(std::string positionPath, std::string densityPath)
{
	FILE* file = NULL;
	errno_t err;

	err = fopen_s(&file, positionPath.c_str(), "rb");
	if (err != 0) {
		printf("failed to open File [%s]\n", positionPath.c_str());
		return false;
	}

	fseek(file, 0, SEEK_END);
	long fileSize = ftell(file);
	fseek(file, 0, SEEK_SET);

	printf("file size : %ld\n", fileSize/(3*sizeof(float)));

	particles = new Particle[fileSize / (3 * sizeof(float))];

	particleSize = fileSize / (3 * sizeof(float));
	for (int i = 0; i < fileSize/(3*sizeof(float)); ++i) {
		vec3 tmpPosition;
		fread(&(particles[i].position.x), sizeof(float), 1, file);
		fread(&(particles[i].position.y), sizeof(float), 1, file);
		fread(&(particles[i].position.z), sizeof(float), 1, file);
	}

	fclose(file);

	FILE* file2 = NULL;
	errno_t err2;
	err2 = fopen_s(&file2, densityPath.c_str(), "rb");
	if (err2 != 0) {
		printf("failed to open File [%s]\n", densityPath.c_str());
		return false;
	}

	fseek(file2, 0, SEEK_END);
	long fileSize2 = ftell(file2);
	fseek(file2, 0, SEEK_SET);

	printf("file size : %ld\n", fileSize / (3 * sizeof(float)));

	for (int i = 0; i < fileSize / (3 * sizeof(float)); ++i) {
		fread(&(particles[i].density), sizeof(float), 1, file2);
	}
	float minDensity = particles[0].density;
	float maxDensity = particles[0].density;
	for (int i = 0; i < particleSize; ++i)
	{
		if (particles[i].density < minDensity) minDensity = particles[i].density;
		if (particles[i].density > maxDensity) maxDensity = particles[i].density;
	}

	printf("minDensity = %f\n", minDensity);
	printf("maxDensity = %f\n", maxDensity);

	fclose(file2);

	return true;
}

bool MarchingCube::make_polygon_with_particles(std::vector<vec3> vertices, float isoValue)
{
	particles = new Particle[vertices.size()];
	particleSize = vertices.size();

	for (int i = 0; i < vertices.size(); ++i)
	{
		particles[i].position = vertices[i];
		particles[i].density = 94.0;
	}
	generate_grid();

	printf("x Size : %d\ty Size : %d\tz Size : %d\n", axisX, axisY, axisZ);

	if (put_density_into_cell()) printf("put density into cell \n");

	alloc_device_memory();

	compute_cell_bit(isoValue);

	make_triangle(isoValue);

	make_triangle_arr();

	return true;
}

bool MarchingCube::make_polygon_with_particles(float isoValue)
{
	if (particleSize == 0) {
		printf("[ERR] No particles\n");
		return false;
	}
	generate_grid();

	printf("x Size : %d\ty Size : %d\tz Size : %d\n", axisX, axisY, axisZ);

	if (put_density_into_cell()) printf("put density into cell \n");

	alloc_device_memory();

	compute_cell_bit(isoValue);

	make_triangle(isoValue);

	make_triangle_arr();

	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA:ERROR:cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	}
	else {
		printf("CUDA Success\n");
	}
	printf("inside make_cell_triangle\n");
	return true;
}

bool MarchingCube::generate_grid()
{
	find_grid_minmax();

	vec3 tmpVertex = maxVertex - minVertex;
	gridSize = std::min(tmpVertex.x, std::min(tmpVertex.y, tmpVertex.z)) / 40;
	
	axisX = (int(tmpVertex.x / gridSize) + 3);
	axisY = (int(tmpVertex.y / gridSize) + 3);
	axisZ = (int(tmpVertex.z / gridSize) + 3);

	printf("%d, %d, %d\n", axisX, axisY, axisZ);
	initialize_cell();
	printf("size of cell array = %d\n", sizeof(Cell) * axisX * axisY * axisZ);

	return true;
}

bool MarchingCube::put_density_into_cell()
{
	for (int i = 0; i < particleSize; ++i)
	{
		h_data.cells[(int((particles[i].position.x - minVertex.x) / gridSize)*axisY*axisZ)+(int((particles[i].position.y - minVertex.y) / gridSize)*axisZ) + int((particles[i].position.z - minVertex.z) / gridSize)].particleCnt++;
	}

	for (int i = 0; i < particleSize; ++i)
	{
		h_data.cells[(int((particles[i].position.x - minVertex.x) / gridSize) * axisY * axisZ) + (int((particles[i].position.y - minVertex.y) / gridSize) * axisZ) + int((particles[i].position.z - minVertex.z) / gridSize)].density += particles[i].density / h_data.cells[(int((particles[i].position.x - minVertex.x) / gridSize) * axisY * axisZ) + (int((particles[i].position.y - minVertex.y) / gridSize) * axisZ) + int((particles[i].position.z - minVertex.z) / gridSize)].particleCnt;
	}
	return true;
}

bool MarchingCube::initialize_cell()
{
	h_data.cells = new Cell[axisX * axisY * axisZ];

	for (int i = 0; i < axisX; ++i)
	{
		for (int j = 0; j < axisY; ++j)
		{
			for (int k = 0; k < axisZ; ++k)
			{
				h_data.cells[(i * axisY * axisZ) + (j * axisZ) + k].coordinate = vec3{
					minVertex.x + (gridSize / 2) + gridSize * i,
					minVertex.y + (gridSize / 2) + gridSize * j,
					minVertex.z + (gridSize / 2) + gridSize * k
				};

				h_data.cells[(i * axisY * axisZ) + (j * axisZ) + k].set_vertex_with_coordinate(gridSize);
			}
		}
	}

	//h_triangles = new Triangle[axisX * axisY * axisZ];

	return true;
}

bool MarchingCube::get_vertices_by_vtk(std::string filepath)
{
	std::string line;
	char* testLine;
	std::ifstream file(filepath);

	if (!file.is_open()) {
		std::cerr << "파일을 열 수 없습니다." << std::endl;
		return 1;
	}

	while (std::getline(file, line)) {
		if (line.find("POLYDATA") != std::string::npos) {
			std::cout << line << std::endl;
			break;
		}
	}
	std::string tmpStr;
	file >> tmpStr;
	int numFloats;
	file >> numFloats;
	file >> tmpStr;
	float* tmpFloat;

	std::cout << numFloats << std::endl;

	tmpFloat = new float[numFloats];

	for (int i = 0; i < numFloats; ++i)
	{
		tmpFloat[i] = 0.0;
		file >> tmpFloat[i];
		std::cout << tmpFloat[i] << std::endl;
	}
	/*
	std::cout << numFloats << std::endl;

	while (std::getline(file, line)) {
		if (line.find("POINTS") != std::string::npos) {
			std::cout << line << std::endl;
			break;
		}
	}
	*/
	//std::getline(file, line);
	//std::cout << line << std::endl;

	/*
	std::vector<unsigned char> floatData(numFloats * sizeof(unsigned char));

	file.read(reinterpret_cast<char*>(floatData.data()), numFloats * sizeof(unsigned char));

	//file.read()
	for (unsigned char value : floatData) {
		std::cout << static_cast<int>(value) << " ";
	}
	
	*/

	//std::vector<float> floatData(numFloats);
	//file.read(reinterpret_cast<char*>(floatData.data()), numFloats * sizeof(float));

	//for (float value : floatData) {
	//	std::cout << value << std::endl;
	//}

	file.close();
	
	/*
	if (file.is_open()) {
		while (std::getline(file, line)) {
			if (isInsidePoint) {
				file.read(testLine, 282900 * sizeof(float));
				printf("%s\n", testLine);
			}
			if (line.find("POINTS") != std::string::npos) {
				isInsidePoint = true;
			}
		}
	}
	*/
	return true;
}

bool MarchingCube::find_grid_minmax()
{
	if (particleSize == 0) {
		printf("[ERR] No particles\n");
		return false;
	}
	minVertex = particles[0].position;
	maxVertex = particles[0].position;


	for (int i = 0; i < particleSize; ++i) {
		if (minVertex.x > particles[i].position.x) minVertex.x = particles[i].position.x;
		if (minVertex.y > particles[i].position.y) minVertex.y = particles[i].position.y;
		if (minVertex.z > particles[i].position.z) minVertex.z = particles[i].position.z;

		if (maxVertex.x < particles[i].position.x) maxVertex.x = particles[i].position.x;
		if (maxVertex.y < particles[i].position.y) maxVertex.y = particles[i].position.y;
		if (maxVertex.z < particles[i].position.z) maxVertex.z = particles[i].position.z;
	}
	printf("min : (%f, %f, %f)\n", minVertex.x, minVertex.y, minVertex.z);
	printf("max : (%f, %f, %f)\n", maxVertex.x, maxVertex.y, maxVertex.z);
	return true;
}

void MarchingCube::print_txt(std::string filepath)
{
	FILE* file = NULL;

	fopen_s(&file, filepath.c_str(), "wb");

	for (int i = 0; i < h_data.triangles.size(); ++i) {

		fwrite(&h_data.triangles[i].t1.x, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t1.y, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t1.z, sizeof(float), 1, file);

		fwrite(&h_data.triangles[i].t2.x, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t2.y, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t2.z, sizeof(float), 1, file);

		fwrite(&h_data.triangles[i].t3.x, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t3.y, sizeof(float), 1, file);
		fwrite(&h_data.triangles[i].t3.z, sizeof(float), 1, file);
	}

	fclose(file);
}

void MarchingCube::print_vtu(std::string filepath)
{
	FILE* file = NULL;

	fopen_s(&file, filepath.c_str(), "wb");

	std::string txt = "<VTKFile type=\"UnstructuredGrid\" version=\"1.0\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<UnstructuredGrid>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);
	txt = "<Piece NumberOfPoints=\"" + std::to_string(h_data.triangles.size() * 3) + "\" NumberOfCells=\"" + std::to_string(h_data.triangles.size()) + "\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);
	txt = "<Points>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);
	txt = "<DataArray type=\"Float64\" NumberOfComponents=\"" + std::to_string(3) + "\" format=\"ascii\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);
	txt = "";
	for (int i = 0; i < h_data.triangles.size(); ++i) {
		txt += (std::to_string(h_data.triangles[i].t1.x) + " " + std::to_string(h_data.triangles[i].t1.y) + " " + std::to_string(h_data.triangles[i].t1.z) + "\n");
		txt += (std::to_string(h_data.triangles[i].t2.x) + " " + std::to_string(h_data.triangles[i].t2.y) + " " + std::to_string(h_data.triangles[i].t2.z) + "\n");
		txt += (std::to_string(h_data.triangles[i].t3.x) + " " + std::to_string(h_data.triangles[i].t3.y) + " " + std::to_string(h_data.triangles[i].t3.z) + "\n");
	}

	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</DataArray>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</Points>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<Cells>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<DataArray type=\"Int32\" Name=\"offsets\" format=\"ascii\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "";

	for (int i = 0; i < h_data.triangles.size(); ++i)
	{
		txt += std::to_string(i * 3) + " ";
	}
	txt += "\n";

	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</DataArray>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "";
	for (int i = 0; i < h_data.triangles.size()*3; ++i)
	{
		if (i % 3 == 2) txt += (std::to_string(i) + "\n");
		else txt += (std::to_string(i) + " ");
	}
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</DataArray>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<DataArray type=\"UInt8\" Name=\"types\" format=\"ascii\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "";
	for (int i = 0; i < h_data.triangles.size(); ++i)
	{
		txt += "5 ";
	}
	txt += "\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</DataArray>\n</Cells>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "<PointData>\n<DataArray type = \"Float32\" Name=\"Density\" format=\"ascii\">\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "";

	for (int i = 0; i < h_data.triangles.size(); ++i)
	{
		txt += std::to_string(h_data.triangles[i].density[0]) + " ";
		txt += std::to_string(h_data.triangles[i].density[1]) + " ";
		txt += std::to_string(h_data.triangles[i].density[2]) + " ";
	}
	txt += "\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</DataArray>\n</PointData>\n";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);

	txt = "</Piece>\n</UnstructuredGrid>\n</VTKFile>";
	fwrite(txt.c_str(), sizeof(char), txt.size(), file);
	fclose(file);
}

void MarchingCube::set_density() {

}
void MarchingCube::set_vertices()
{

}

void MarchingCube::write_binary(std::string txt)
{

}