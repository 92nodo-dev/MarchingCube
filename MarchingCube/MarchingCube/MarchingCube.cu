#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "MarchingCube.cuh"

__global__ void compute_bit(Cell*** cell) {
	printf("test\n");
}

void compute_cell_bit(Cell**** cells)
{
	Cell*** d_cells;

	hipMalloc(&d_cells, sizeof(cells));

	hipMemcpy(d_cells, cells, sizeof(cells), hipMemcpyHostToDevice);

	compute_bit << <1, 15 >> > (d_cells);

	hipFree(d_cells);
}

void free_gpu_memory()
{

}