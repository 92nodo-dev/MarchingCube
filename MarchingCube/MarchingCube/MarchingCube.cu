#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "MarchingCube.cuh"

__global__ void compute_bit(Cell* cell, int x, int y, int z) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int zIndex = int(idx % z);
	int yIndex = int((idx / z) % y);
	int xIndex = int(idx / (y * z));
	//printf("xIndex : %d\tyIndex : %d\tzIndex : %d\n", xIndex, yIndex, zIndex);
	//printf("yIndex : %d\t", yIndex);
	//printf("test : %f\n", cell[idx].density);
	//printf("x : %f\ty : %f\tz : %f\n", cell[xIndex][yIndex][zIndex].coordinate.x, cell[xIndex][yIndex][zIndex].coordinate.y, cell[xIndex][yIndex][zIndex].coordinate.z);

	//printf("test : %d\n", idx);
}

/*
__global__ void compute_bit(Cell*** cell, int x, int y, int z) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int zIndex = int(idx % z);
	int yIndex = int((idx / z) % y);
	int xIndex = int(idx / (y * z));
	printf("xIndex : %d\tyIndex : %d\tzIndex : %d\n", xIndex, yIndex, zIndex);
	//printf("yIndex : %d\t", yIndex);
	printf("test : %d\n", cell[0][0][0].particleCnt);
	//printf("x : %f\ty : %f\tz : %f\n", cell[xIndex][yIndex][zIndex].coordinate.x, cell[xIndex][yIndex][zIndex].coordinate.y, cell[xIndex][yIndex][zIndex].coordinate.z);

	//printf("test : %d\n", idx);
}
*/

void compute_cell_bit(Cell*** cells, int axisX, int axisY, int axisZ)
{
	//Cell*** d_cells;
	Cell* d_cells;
	int d_idx = 0;
	hipMalloc((void**)&d_cells, axisX * axisY * axisZ* sizeof(Cell));

	for (int i = 0; i < axisX; ++i)
	{
		for (int j = 0; j < axisY; ++j)
		{
			for (int k = 0; k < axisZ; ++k)
			{
				//printf("density : %f\n", cells[i][j][k].density);
				hipMemcpy(&d_cells[d_idx], &cells[i][j][k], sizeof(Cell), hipMemcpyHostToDevice);
				d_idx++;
			}
		}
	}
	//hipMemcpy(d_cells, cells, axisX * axisY * axisZ * sizeof(Cell), hipMemcpyDeviceToDevice);
	
	compute_bit << <1, axisX*axisY*axisZ >> > (d_cells, axisX, axisY, axisZ);

	hipFree(d_cells);
}